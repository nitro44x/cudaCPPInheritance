#include "hip/hip_runtime.h"
#include "GPUPrinter.cuh"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>

HOSTDEVICE void GPUPrinter::print_your_thing() const
{
    printf("Hello from gpu\n");
}

template <typename T, typename D>
__global__ void newKernel(T ** obj)
{
    *obj = new D;
}
__global__ void printKernel(ISimplePrinter ** printer)
{
    (*printer)->print_your_thing();
}
template <typename T>
__global__ void delKernel(T ** obj)
{
    delete (*obj);
    *obj = nullptr;
}

int printFromGPU()
{
    hipError_t err = hipSuccess;
    ISimplePrinter ** printer_dev = nullptr;
    err = hipMalloc((void**)&printer_dev, sizeof(ISimplePrinter**));
    newKernel<ISimplePrinter, GPUPrinter><<<1,1>>>(printer_dev);
    err = hipDeviceSynchronize();
    printKernel<<<1,1>>>(printer_dev);
    err = hipDeviceSynchronize();
    delKernel<<<1,1>>>(printer_dev);
    err = hipDeviceSynchronize();
    err = hipFree(printer_dev);
    return hipSuccess == err;
}

