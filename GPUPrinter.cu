#include "hip/hip_runtime.h"
#include "GPUPrinter.cuh"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>

using printer_handle = ISimplePrinter**;

HOSTDEVICE void GPUPrinter::print_your_thing() const
{
    printf("Hello from gpu\n");
}

__global__ void newKernel(printer_handle obj)
{
    *obj = new GPUPrinter;
}
__global__ void printKernel(printer_handle printer)
{
    (*printer)->print_your_thing();
}
__global__ void delKernel(printer_handle obj)
{
    delete (*obj);
    *obj = nullptr;
}

int printFromGPU()
{

    hipError_t err = hipSuccess;
    printer_handle printer_dev = nullptr;
    err = hipMalloc((void**)&printer_dev, sizeof(printer_handle));
    newKernel<<<1,1>>>(printer_dev);
    err = hipDeviceSynchronize();
    printKernel<<<1,1>>>(printer_dev);
    err = hipDeviceSynchronize();
    delKernel<<<1,1>>>(printer_dev);
    err = hipDeviceSynchronize();
    err = hipFree(printer_dev);
    return hipSuccess == err;
}

